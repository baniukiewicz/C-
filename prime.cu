#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <chrono> 
#include <omp.h>
using namespace std::chrono;
#define THREADS_PER_BLOCK 1024
__global__ void prm(unsigned long long int* a, int* b, int* c, int* d) {
	unsigned long long int i = threadIdx.x + blockIdx.x * blockDim.x;
	
	if (*a % 2 == 0 && *a != 2)
	{
		*b = *d;
	}
	else 
	{
		if (*a % i == 0 && i != 1 && i <= *c)
		{
			*b = *d;
		}
	}
}


int main(void) {
	int b, c, d, e, f;
	unsigned long long int a;
	unsigned long long int* d_a;
	int* d_b, * d_c, * d_d;
	int size_m = sizeof(int);
	unsigned long long int size_d = sizeof(unsigned long long int);

	hipMalloc((void**)&d_a, size_d);
	hipMalloc((void**)&d_b, size_m);
	hipMalloc((void**)&d_c, size_m);
	hipMalloc((void**)&d_d, size_m);

	std::cout << "podaj liczbe " << std::endl;
	std::cin >> a;

	b = 0;
	d = 1;
	e = 0;
	f = 1;

	c = ceil(sqrt(a));
	auto start1 = high_resolution_clock::now();
	if (a % 2 == 0 && a != 2)
		e = f;
	else {
		for (int i = 3; i <= c; i += 2)
		{
			if (a % i == 0)
			{
				e = f;
			}
		};
	};
	
	auto stop1 = high_resolution_clock::now();
	auto duration1 = duration_cast<nanoseconds>(stop1 - start1);

	if (!e)
	{
		std::cout << "CPU: number is prime, computed in = " << duration1.count() << "ns" << std::endl;
	}
	if (e)
	{
		std::cout << "CPU: number is not prime, computed in = " << duration1.count() << "ns" << std::endl;
	}

	e = 0;
	auto start3 = high_resolution_clock::now();
	

		
		if (a % 2 == 0 && a != 2)
			e = f;
		else {
			#pragma omp parallel shared(a,c,e,f)
			{
			#pragma omp for
				for (int i = 3; i <= c; i += 2)
				{
					if (a % i == 0)
					{
						e = f;
					}
				};
			}
		};
	
	auto stop3 = high_resolution_clock::now();
	auto duration3 = duration_cast<nanoseconds>(stop3 - start3);

	if (!e)
	{
		std::cout << "CPU OpenMP: number is prime, computed in = " << duration3.count() << "ns" << std::endl;
	}
	if (e)
	{
		std::cout << "CPU OpenMP: number is not prime, computed in = " << duration3.count() << "ns" << std::endl;
	}

	hipMemcpy(d_a, &a, size_d, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size_m, hipMemcpyHostToDevice);
	hipMemcpy(d_c, &c, size_m, hipMemcpyHostToDevice);
	hipMemcpy(d_d, &d, size_m, hipMemcpyHostToDevice);

	auto start2 = high_resolution_clock::now();
	int N = 512;

	prm << <N, THREADS_PER_BLOCK >> > (d_a, d_b, d_c, d_d);

	auto stop2 = high_resolution_clock::now();
	auto duration2 = duration_cast<nanoseconds>(stop2 - start2);

	hipMemcpy(&b, d_b, size_m, hipMemcpyDeviceToHost);

	if (!b) 
	{ 
		std::cout << "GPU: number is prime, computed in = " << duration2.count() << "ns" << std::endl; 
	}
	if (b) 
	{ 
		std::cout << "GPU: number is not prime, computed in = " << duration2.count() << "ns" << std::endl; 
	}
	
	hipFree(d_a); hipFree(d_b); hipFree(d_c);

	return 0;
}
